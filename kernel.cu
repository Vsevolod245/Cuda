#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include <hip/hip_math_constants.h>

#define N (256*256)

__global__ void kernel(float* data)
{
	int idx =(int)(blockIdx.x * blockDim.x + threadIdx.x);
	data[idx] = cosf(idx * HIP_PI_F / (2.0 * N));
}

int main(int argc, char* argv[])
{
	float a[N];
	float* dev = NULL;
	hipMalloc((void**)&dev, N * sizeof(float));
	kernel << <dim3((N / 512), 1), dim3(512, 1) >> > (dev);
	hipMemcpy(a, dev, N * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(dev);
	for (int idx = 0; idx < N; idx++)
		printf("Cosinus of %f = %.5f\n", 90.0 * idx / N, a[idx]);
	return 0;
}